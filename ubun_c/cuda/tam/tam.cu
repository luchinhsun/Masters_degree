#include "hip/hip_runtime.h"
#include "head.h"

extern float *d_a_tam, *d_b_tam,*d_c_tam;
extern float *d_c_new_tam,*d_d_new_tam;
extern float *d_Ax, *d_Ay;
extern float *d_b, *d_ut, *d_Vt;

extern float *d_V1, *d_V2;
//extern float *d_b, *d_ut, *d_Vt;

__global__ void GPU_adi_x(float *d_V1, float *d_b, int j){
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        
	if(i<Np){
        if(j==0){
                d_b[i] = d_V1[i*Np+j] + r*(-d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }else if(j==Np-1){
                d_b[i] = d_V1[i*Np+j] + r*(d_V1[i*Np+j-1] - d_V1[i*Np+j]);
        }else{
                d_b[i] = d_V1[i*Np+j] + (r/2)*(d_V1[i*Np+j-1] - 2*d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }
	}

}

__global__ void GPU_getV2(float *d_V2, float *d_Vt, int j){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i<Np){
		d_V2[i*Np+j] = d_Vt[i];
	}

}

__global__ void GPU_adi_y(float *d_V2, float *d_b, int i){
        int j = blockDim.x * blockIdx.x + threadIdx.x;

	if(j<Np){
	if(i==0){
        	d_b[j] = d_V2[i*Np+j] + (r/2)*(-2*d_V2[i*Np+j] + 2*d_V2[(i+1)*Np+j]);
        }else if(i==Np-1){
                d_b[j] = d_V2[i*Np+j] + (r/2)*(2*d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j]);
        }else{
                d_b[j] = d_V2[i*Np+j] + (r/2)*(d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j] + d_V2[(i+1)*Np+j]);
        }
	}
}

__global__ void GPU_getV1(float *d_V1, float *d_ut, int i){
        int j = blockDim.x * blockIdx.x + threadIdx.x;

        if(j<Np){
                d_V1[i*Np+j] = d_ut[j];
        }

}

__global__ void GPU_tam(float *A, float *d_a_tam, float *d_b_tam, float *d_c_tam, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i==0){
		d_a_tam[0] = 0.0;
		d_c_tam[n-1] = 0.0;
	}
	__syncthreads();

	if(i<n-1){
		d_c_tam[i] = A[i*n+i+1];
		d_a_tam[i+1] = A[(i+1)*n+i];
	}

	if(i<n){
		d_b_tam[i] = A[i*n+i];
	}
}

__global__ void GPU_tam2(float *d_a_tam, float *d_b_tam, float *d_c_tam, float *d_c_new_tam, float *d_d_new_tam, float *d, float *x, int n){
	int i;
	d_c_new_tam[0] = d_c_tam[0]/d_b_tam[0];
        for(i=1;i<n-1;i++){
                d_c_new_tam[i] = d_c_tam[i]/(d_b_tam[i]-d_a_tam[i]*d_c_new_tam[i-1]);
        }

        d_d_new_tam[0] = d[0]/d_b_tam[0];
        for(i=1;i<n;i++){
                d_d_new_tam[i] = (d[i]-d_a_tam[i]*d_d_new_tam[i-1])/(d_b_tam[i]-d_a_tam[i]*d_c_new_tam[i-1]);
        }

        x[n-1] = d_d_new_tam[n-1];
        for(i=n-2;i>-1;i--){
                x[i] = d_d_new_tam[i]-d_c_new_tam[i]*x[i+1];
        }

}

void tam(){
	int tpb = 256;
	int bpg = (Np*Np+tpb-1)/tpb;
	int bpg1 = (Np+tpb-1)/tpb;

	//GPU_tam<<<bpg, tpb>>>(d_Ax, d_a_tam, d_b_tam, d_c_tam, Np);
	int j;
	for(j=0;j<Np;j++){
		GPU_adi_x<<<bpg, tpb>>>(d_V1, d_b, j);
		GPU_tam<<<bpg, tpb>>>(d_Ax, d_a_tam, d_b_tam, d_c_tam, Np);
		GPU_tam2<<<1, 1>>>(d_a_tam, d_b_tam, d_c_tam, d_c_new_tam, d_d_new_tam, d_b, d_Vt, Np);
		GPU_getV2<<<bpg1, tpb>>>(d_V2, d_Vt, j);
	}
}

void tam2(){
        int tpb = 256;
        int bpg = (Np*Np+tpb-1)/tpb;
	int bpg1 = (Np+tpb-1)/tpb;

        //GPU_tam<<<bpg, tpb>>>(d_Ay, d_a_tam, d_b_tam, d_c_tam, Np);
	int i;
	for(i=0;i<Np;i++){
		GPU_adi_y<<<bpg, tpb>>>(d_V2, d_b, i);
		GPU_tam<<<bpg, tpb>>>(d_Ay, d_a_tam, d_b_tam, d_c_tam, Np);
        	GPU_tam2<<<1, 1>>>(d_a_tam, d_b_tam, d_c_tam, d_c_new_tam, d_d_new_tam, d_b, d_ut, Np);
		GPU_getV1<<<bpg1, tpb>>>(d_V1, d_ut, i);
	}
}

