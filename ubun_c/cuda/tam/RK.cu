#include "hip/hip_runtime.h"
#include "head.h"

extern float *d_F, *d_x, *d_y;
extern float *d_V2;
extern float *d_V_tmp;
extern float *d_t;

__global__ void GPU_fsource(float *d_V_tmp, float *d_t, float *d_x, float *d_y, float *d_F){
        int k = blockDim.x * blockIdx.x + threadIdx.x;
        int j;

        if(k<Np){
                for(j=0;j<Np;j++){
                        d_F[k*Np+j] = exp(-2.0*d_t[0])*cos(M_PI*d_x[k])*
                                                cos(M_PI*d_y[j])*(2.0*M_PI*M_PI)
                                                        -2.0*d_V_tmp[k*Np+j];
                }
        }
}

__global__ void GPU_RKa(float *d_V_tmp, float *d_V2, float *d_F, float *d_t){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i<Np*Np){
		d_V_tmp[i] = d_V2[i] + (1.0/2.0)*dt*d_F[i];
	}

	if(i==0)	d_t[0] = d_t[0]+dt/2.0;
}

__global__ void GPU_RKb(float *d_V2, float *d_F, float *d_t){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i<Np*Np){
		d_V2[i] = d_V2[i] + dt*d_F[i];
        }

	if(i==0)	d_t[0] = d_t[0]+dt/2.0;
}

void RK(){
	int tpb = 256;
        int bpg = (Np*Np+tpb-1)/tpb;

	GPU_RKa<<<bpg, tpb>>>(d_V_tmp, d_V2, d_F, d_t);
        GPU_fsource<<<bpg, tpb>>>(d_V_tmp, d_t, d_x, d_y, d_F);
	GPU_RKb<<<bpg, tpb>>>(d_V2, d_F, d_t);
}

