#include "hip/hip_runtime.h"
#include "head.h"

int k_t = (tfinal/dt);

float *Ax, *Ay, *V1, *V2, *W, *F, *V_tmp, *W_tmp, *ue;
float *b, *x, *y, *ut, *Vt;
float *t;

//GPU variable
float *d_Ax, *d_Ay, *d_V1, *d_V2;
float *d_b, *d_ut, *d_Vt;

//GPU tam variable
float *d_a_tam, *d_b_tam,*d_c_tam;
float *d_c_new_tam,*d_d_new_tam;

//GPU RK variable
float *d_F, *d_x, *d_y, *d_V_tmp;
float *d_t;

clock_t start;
clock_t end;
float time_used;

void Allocate(){

	size_t size;
	size = Np*Np*sizeof(float);

	Ax = (float*)malloc(size);
	Ay = (float*)malloc(size);
	V1 = (float*)malloc(size);
	V2 = (float*)malloc(size);
	W = (float*)malloc(size);
	F = (float*)malloc(size);
	V_tmp = (float*)malloc(size);
	W_tmp = (float*)malloc(size);
	ue = (float*)malloc(size);

	hipError_t Error;

        Error = hipMalloc((void**)&d_Ax, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_Ax) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_Ay, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_Ay) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_V1, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_V1) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_V2, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_V2) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_V_tmp, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_V_tmp) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_F, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_F) = %s\n", hipGetErrorString(Error));

	size = Np*sizeof(float);

	b = (float*)malloc(size);
	x = (float*)malloc(size);
	y = (float*)malloc(size);
	ut = (float*)malloc(size);
	Vt = (float*)malloc(size);

        Error = hipMalloc((void**)&d_b, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_b) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_x, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_x) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_y, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_y) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_ut, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_ut) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_Vt, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_Vt) = %s\n", hipGetErrorString(Error));

	Error = hipMalloc((void**)&d_a_tam, size);
	if(Error != hipSuccess)
	printf("CUDA error(malloc d_a_tam) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_b_tam, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_b_tam) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_c_tam, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_c_tam) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_c_new_tam, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_c_new_tam) = %s\n", hipGetErrorString(Error));
        Error = hipMalloc((void**)&d_d_new_tam, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_d_new_tam) = %s\n", hipGetErrorString(Error));

	size = 1*sizeof(float);
	t = (float*)malloc(size);
        Error = hipMalloc((void**)&d_t, size);
        if(Error != hipSuccess)
        printf("CUDA error(malloc d_t) = %s\n", hipGetErrorString(Error));
}

void Save_Result(){

	FILE *pFile;
	int i,j;
	int index;
	int n;
	n = Np;
	pFile = fopen("V1.txt","w+");
	// Save the matrix A
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			index = i*n + j;
			fprintf(pFile, "%g", V1[index]);
			if (j == (n-1)) {
        			fprintf(pFile, "\n");
            		}else{
            			fprintf(pFile, "\t");
            		}
    		}
	}
    	fclose(pFile);
/*
    pFile = fopen("F.txt","w+");
    // Save the matrix A
    for (i = 0; i < n; i++) {
    	for (j = 0; j < n; j++) {
        	index = i*n + j;
            fprintf(pFile, "%g", F[index]);
            if (j == (n-1)) {
        		fprintf(pFile, "\n");
            }else{
            	fprintf(pFile, "\t");
            }
    	}
	}
    fclose(pFile);
    pFile = fopen("V_tmp.txt","w+");
    // Save the matrix A
    for (i = 0; i < n; i++) {
    	for (j = 0; j < n; j++) {
        	index = i*n + j;
            fprintf(pFile, "%g", V_tmp[index]);
            if (j == (n-1)) {
        		fprintf(pFile, "\n");
            }else{
            	fprintf(pFile, "\t");
            }
    	}
	}
    fclose(pFile);
    pFile = fopen("b.txt","w+");
    for (i = 0; i < n; i++) {
        fprintf(pFile, "%g", b[i]);
        fprintf(pFile, "\t");
    }
    fclose(pFile);
    pFile = fopen("x.txt","w+");
    for (i = 0; i < n; i++) {
        fprintf(pFile, "%g", x[i]);
        fprintf(pFile, "\t");
    }
    fclose(pFile);
	*/
}

void Free(){
	free(Ax);free(Ay);free(V1);free(V2);
	free(W);free(F);free(V_tmp);free(W_tmp);free(ue);
	free(b);free(x);free(y);free(ut);free(Vt);
	free(t);

	hipFree(d_Ax);hipFree(d_Ay);hipFree(d_V1);hipFree(d_V2);
	hipFree(d_V_tmp);hipFree(d_F);
	hipFree(d_t);
	hipFree(d_b);hipFree(d_x);hipFree(d_y);hipFree(d_Vt);hipFree(d_ut);
	hipFree(d_a_tam);hipFree(d_b_tam);hipFree(d_c_tam);
	hipFree(d_c_new_tam);hipFree(d_d_new_tam);
}

void Send_to_Device(){
	start = clock();

	hipError_t Error;
	size_t size;
	size = Np*Np*sizeof(float);
	Error = hipMemcpy(d_V1, V1, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy V1->d_V1) = %s\n",hipGetErrorString(Error));
/*
        Error = hipMemcpy(d_V2, V2, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy V2->d_V2) = %s\n",hipGetErrorString(Error));
*/
	Error = hipMemcpy(d_Ax, Ax, size, hipMemcpyHostToDevice);
	if (Error != hipSuccess)
	printf("CUDA error(copy Ax->d_Ax) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_Ay, Ay, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy Ay->d_Ay) = %s\n",hipGetErrorString(Error));

	Error = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy F->d_F) = %s\n",hipGetErrorString(Error));
	
	size = Np*sizeof(float);
	Error = hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy x->d_x) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy y->d_y) = %s\n",hipGetErrorString(Error));

	size = 1*sizeof(float);
        Error = hipMemcpy(d_t, t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy t->d_t) = %s\n",hipGetErrorString(Error));

/*
	size = Np*sizeof(float);
        Error = hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy b->d_b) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_ut, ut, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy ut->d_ut) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_Vt, Vt, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy Vt->d_Vt) = %s\n",hipGetErrorString(Error));
*/
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
	size = Np*Np*sizeof(float);
	Error = hipMemcpy(V1, d_V1, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V1->V1) = %s\n",hipGetErrorString(Error));
/*
	Error = hipMemcpy(V2, d_V2, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V2->V2) = %s\n",hipGetErrorString(Error));

	Error = hipMemcpy(V_tmp, d_V_tmp, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V_tmp->V_tmp) = %s\n",hipGetErrorString(Error));

        Error = hipMemcpy(Ax, d_Ax, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Ax->Ax) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(Ay, d_Ay, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Ay->Ay) = %s\n",hipGetErrorString(Error));
        size = Np*sizeof(float);
        Error = hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_b->b) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(ut, d_ut, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_ut->ut) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(Vt, d_Vt, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vt->Vt) = %s\n",hipGetErrorString(Error));
*/

	end = clock();
        time_used = (float)(end - start)/ CLOCKS_PER_SEC;
        printf("\ntime in cu = %f\n",time_used);
}

