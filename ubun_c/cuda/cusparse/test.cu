#include "hip/hip_runtime.h"
#include "head.h"

extern float *b, *ut, *Vt;
extern float *d_V1, *d_V2;
extern float *d_b, *d_ut, *d_Vt;

__global__ void GPU_adi_x(float *d_V1, float *d_b, int j){
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i<Np){
        if(j==0){
                d_b[i] = d_V1[i*Np+j] + r*(-d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }else if(j==Np-1){
                d_b[i] = d_V1[i*Np+j] + r*(d_V1[i*Np+j-1] - d_V1[i*Np+j]);
        }else{
                d_b[i] = d_V1[i*Np+j] + (r/2)*(d_V1[i*Np+j-1] - 2*d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }
        }
}

__global__ void GPU_getV2(float *d_V2, float *d_Vt, int j){
        int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i<Np){
                d_V2[i*Np+j] = d_Vt[i];
        }

}

__global__ void GPU_adi_y(float *d_V2, float *d_b, int i){
        int j = blockDim.x * blockIdx.x + threadIdx.x;

        if(j<Np){
        if(i==0){
                d_b[j] = d_V2[i*Np+j] + (r/2)*(-2*d_V2[i*Np+j] + 2*d_V2[(i+1)*Np+j]);
        }else if(i==Np-1){
                d_b[j] = d_V2[i*Np+j] + (r/2)*(2*d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j]);
        }else{
                d_b[j] = d_V2[i*Np+j] + (r/2)*(d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j] + d_V2[(i+1)*Np+j]);
        }
        }
}

__global__ void GPU_getV1(float *d_V1, float *d_ut, int i){
        int j = blockDim.x * blockIdx.x + threadIdx.x;

        if(j<Np){
                d_V1[i*Np+j] = d_ut[j];
        }

}
void ADI1(int j){
        int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_adi_x<<<bpg, tpb>>>(d_V1, d_b, j);
}
void ADI1_2(int j){
	int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_getV2<<<bpg, tpb>>>(d_V2, d_Vt, j);
}

void ADI2(int i){
        int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_adi_y<<<bpg, tpb>>>(d_V2, d_b, i);
}

void ADI2_2(int i){
	int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_getV1<<<bpg, tpb>>>(d_V1, d_ut, i);
}

void Send_to_D(){
	size_t size;
	size = Np*sizeof(float);
	hipMemcpy(d_Vt, Vt, size, hipMemcpyHostToDevice);
	hipMemcpy(d_ut, ut, size, hipMemcpyHostToDevice);
}

void Send_to_H(){
	size_t size;
        size = Np*sizeof(float);
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
}
