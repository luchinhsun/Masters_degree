#include "hip/hip_runtime.h"
#include "head.h"
//#include <time.h>
/*
clock_t start_cu;
clock_t end_cu;
float time_used_cu;
*/
//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;
hipsparseMatDescr_t descr=0;
hipsparseMatDescr_t descrL=0;
hipsparseMatDescr_t descrU=0;
cusparseSolveAnalysisInfo_t infoA=0;
cusparseSolveAnalysisInfo_t info_u=0;
int *cooRowIndexHostPtr;
int * cooColIndexHostPtr;
float * cooValHostPtr;
int *cooRowIndex;
int * cooColIndex;
float * cooVal;
float * cooValLU;
float * temp;
float * d_V1_t;
int * csrRowPtr;

float done =1.0;

extern float *d_b, *d_ut, *d_Vt;
extern float *d_V1, *d_V2;

void sparse_Allocate_Memory(){
	//cusparse
	size_t size = nnz*sizeof(int);
	cooRowIndexHostPtr = (int *) malloc(size);
	cooColIndexHostPtr = (int *) malloc(size);
	cooValHostPtr = (float *)malloc(nnz*sizeof(float));

	cooRowIndexHostPtr[0] = 0;cooColIndexHostPtr[0]=0;cooValHostPtr[0]=r+1;
	cooRowIndexHostPtr[1] = 0;cooColIndexHostPtr[1]=1;cooValHostPtr[1]=-r;

	cooRowIndexHostPtr[2] = 1;cooColIndexHostPtr[2]=0;cooValHostPtr[2]=-r/2;
	cooRowIndexHostPtr[3] = 1;cooColIndexHostPtr[3]=1;cooValHostPtr[3]=r+1;
	cooRowIndexHostPtr[4] = 1;cooColIndexHostPtr[4]=2;cooValHostPtr[4]=-r/2;
	int i;
	for(i=5;i<(nnz-3);i=i+3){
		cooRowIndexHostPtr[i] = cooRowIndexHostPtr[i-3]+1;	
		cooColIndexHostPtr[i] = cooColIndexHostPtr[i-3]+1;
		cooRowIndexHostPtr[i+1] = cooRowIndexHostPtr[i];	
		cooColIndexHostPtr[i+1] = cooColIndexHostPtr[i]+1;
		cooRowIndexHostPtr[i+2] = cooRowIndexHostPtr[i+1];	
		cooColIndexHostPtr[i+2] = cooColIndexHostPtr[i+1]+1;
		cooValHostPtr[i]=-r/2;
		cooValHostPtr[i+1]=r+1;
		cooValHostPtr[i+2]=-r/2;
	}
	cooRowIndexHostPtr[nnz-2] = Np-1;cooColIndexHostPtr[nnz-2]=Np-2;cooValHostPtr[nnz-2]=-r;
        cooRowIndexHostPtr[nnz-1] = Np-1;cooColIndexHostPtr[nnz-1]=Np-1;cooValHostPtr[nnz-1]=r+1;

	hipError_t Error;

	Error = hipMalloc((void**)&cooRowIndex, size);
	printf("CUDA error(malloc RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooColIndex, size);
	printf("CUDA error(malloc ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooVal, nnz*sizeof(float));
	printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooValLU, nnz*sizeof(float));
        printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));

	Error = hipMalloc((void**)&temp, Np*Np*sizeof(float));
        printf("CUDA error(malloc temp) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&d_V1_t, Np*Np*sizeof(float));
        printf("CUDA error(malloc d_V1_t) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&csrRowPtr,(Np+1)*sizeof(int));
        printf("CUDA error(malloc csrRowPtr) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	status = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
    	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

    	status = hipsparseCreateMatDescr(&descrU);
    	hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
    	hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
    	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
    	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        status = cusparseCreateSolveAnalysisInfo(&infoA);
        status = cusparseCreateSolveAnalysisInfo(&info_u);
}

void sparse_Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);
	Error = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooColIndex, cooColIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy Val) = %s\n",hipGetErrorString(Error));
}

void cusparse_analysis(){

	status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,Np, csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	/*
        if (status != HIPSPARSE_STATUS_SUCCESS) {
                printf("shit1");
        }
	*/
	status= cusparseScsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, nnz, descr,
                                                cooVal, csrRowPtr, cooColIndex, infoA);
        hipMemcpy(cooValLU, cooVal, nnz*sizeof(float), hipMemcpyDeviceToDevice);
        status = cusparseScsrilu0(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, descr,
                                                cooValLU, csrRowPtr, cooColIndex, infoA);
        status = cusparseScsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, nnz, descrU,
                                                cooVal, csrRowPtr, cooColIndex, info_u);
}

void sparse(){
	status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, Np, &done, descrL,
                                              cooValLU, csrRowPtr, cooColIndex, infoA, d_V2, Np,
						temp, Np);
	status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, Np, &done, descrU,
                                              cooValLU, csrRowPtr, cooColIndex, info_u, temp, Np,
						d_V2, Np);
	/*
	if (status != HIPSPARSE_STATUS_SUCCESS) {
                printf("shit2");
        }
	*/
}

void sparse2(){
        status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, Np, &done, descrL,
                                              cooValLU, csrRowPtr, cooColIndex, infoA, d_V1_t, Np,
						temp, Np);
        status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, Np, Np, &done, descrU,
                                              cooValLU, csrRowPtr, cooColIndex, info_u, temp, Np,
						d_V1_t, Np);
	/*
        if (status != HIPSPARSE_STATUS_SUCCESS) {
                printf("shit4");
        }
	*/
}

void sparse_Free_Memory(){

	status = hipsparseDestroyMatDescr(descr); descr = 0;
	status = hipsparseDestroy(handle); handle = 0;
	status = hipsparseDestroyMatDescr(descrL); descrL = 0;
        status = hipsparseDestroyMatDescr(descrU); descrU = 0;

        if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);
        if (cooColIndexHostPtr) free(cooColIndexHostPtr);
        if (cooValHostPtr) free(cooValHostPtr);
	if (temp) hipFree(temp);
	if (d_V1_t) hipFree(d_V1_t);
        if (csrRowPtr) hipFree(csrRowPtr);
        if (cooRowIndex) hipFree(cooRowIndex);
        if (cooColIndex) hipFree(cooColIndex);
        if (cooVal) hipFree(cooVal);
	if (cooValLU) hipFree(cooValLU);
        if (descr) hipsparseDestroyMatDescr(descr);
	if (descrL) hipsparseDestroyMatDescr(descrL);
	if (descrU) hipsparseDestroyMatDescr(descrU);
        if (handle) hipsparseDestroy(handle);


	cusparseDestroySolveAnalysisInfo(infoA);
        cusparseDestroySolveAnalysisInfo(info_u);

}

__global__ void GPU_adi_x(float *d_V1, float *d_V2){
        int i = blockDim.x * blockIdx.x + threadIdx.x;

	int j;
        if(i<Np){
	for(j=0;j<Np;++j){
        if(j==0){
                d_V2[i*Np+j] = d_V1[i*Np+j] + r*(-d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }else if(j==Np-1){
                d_V2[i*Np+j] = d_V1[i*Np+j] + r*(d_V1[i*Np+j-1] - d_V1[i*Np+j]);
        }else{
                d_V2[i*Np+j] = d_V1[i*Np+j] + (r/2)*(d_V1[i*Np+j-1] - 2*d_V1[i*Np+j] + d_V1[i*Np+j+1]);
        }
        }
	}

}

__global__ void GPU_adi_y(float *d_V2, float *d_V1){
        int j = blockDim.x * blockIdx.x + threadIdx.x;

	int i;
        if(j<Np){
	for(i=0;i<Np;++i){
        if(i==0){
                d_V1[j*Np+i] = d_V2[i*Np+j] + (r/2)*(-2*d_V2[i*Np+j] + 2*d_V2[(i+1)*Np+j]);
        }else if(i==Np-1){
                d_V1[j*Np+i] = d_V2[i*Np+j] + (r/2)*(2*d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j]);
        }else{
                d_V1[j*Np+i] = d_V2[i*Np+j] + (r/2)*(d_V2[(i-1)*Np+j] - 2*d_V2[i*Np+j] + d_V2[(i+1)*Np+j]);
        }
	}
        }
}

__global__ void GPU_trans(float *d_V1, float *d_V1_t){
        int k = blockDim.x * blockIdx.x + threadIdx.x;
        int i;
        int j;

        if(k<Np*Np){
                i = k/Np;
                j = k%Np;
                d_V1[i*Np+j] = d_V1_t[j*Np+i];
        }
}

void ADI1(){
        int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_adi_x<<<bpg, tpb>>>(d_V1, d_V2);
	//start_cu = clock();
        sparse();
	//end_cu = clock();
	//time_used_cu = (float)(end_cu - start_cu)/ CLOCKS_PER_SEC;
	//printf("\n time in sparse kernal = %f\n",time_used_cu);
}

void ADI2(){
        int tpb = 256;
        int bpg = (Np+tpb-1)/tpb;

        GPU_adi_y<<<bpg, tpb>>>(d_V2, d_V1_t);
        sparse2();
	bpg = (Np*Np+tpb-1)/tpb;
        GPU_trans<<<bpg, tpb>>>(d_V1, d_V1_t);
}
