#include "hip/hip_runtime.h"
#include "head.h"

extern float *d_F, *d_x, *d_y;
extern float *d_V1, *d_V2;
extern float *d_t;

__global__ void GPU_fsource(float *d_V2, float *d_t, float *d_x, float *d_y, float *d_F){
        int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i, j;

        if(k<Np*Np){
		i = k/Np;
		j = k%Np;
		d_F[i*Np+j] = exp(-2.0*d_t[0])*cos(M_PI*d_x[i])*
                                                cos(M_PI*d_y[j])*(2.0*M_PI*M_PI)
                                                        -2.0*d_V2[i*Np+j];
        }
}

__global__ void forward(float *d_V1, float *d_V2, float *d_F){
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = k/Np;
        int j = k%Np;

	if(k<Np*Np){
		d_V2[i*Np+j] = d_V1[(i+1)*(Np+2)+j+1] + dt/h1*
                                (d_V1[i*(Np+2)+j+1]+d_V1[(i+2)*(Np+2)+j+1]+d_V1[(i+1)*(Np+2)+j]+d_V1[(i+1)*(Np+2)+j+2]
                                        -4*d_V1[(i+1)*(Np+2)+j+1]) + dt*d_F[i*Np+j];
	}
}

__global__ void update_V1(float *d_V1, float *d_V2, float *d_t){
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i, j;

	if(k<Np*Np){
		i = k/Np;
		j = k%Np;
		d_V1[(i+1)*(Np+2)+j+1] = d_V2[i*Np+j];
	}

	if(k<Np){
		d_V1[k+1] = d_V2[(Np-1)*Np+k];
                d_V1[(Np+1)*(Np+2)+k+1] = d_V2[k];
                d_V1[(k+1)*(Np+2)] = d_V2[k*Np+Np-1];
                d_V1[(k+1)*(Np+2)+Np+1] = d_V2[k*Np];
	}

	if(k==0){
                d_t[0] = d_t[0]+dt;
        }

}

void forward_euler(){
	int tpb = 256;
        int bpg = (Np*Np+tpb-1)/tpb;

	update_V1<<<bpg, tpb>>>(d_V1, d_V2, d_t);
	
	forward<<<bpg, tpb>>>(d_V1, d_V2, d_F);

	GPU_fsource<<<bpg, tpb>>>(d_V2, d_t, d_x, d_y, d_F);
	//hipDeviceSynchronize();
}
