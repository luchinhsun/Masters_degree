#include "head.h"

int k_t = (tfinal/dt);

float *V1, *V2, *F, *ue;
float *x, *y;
float *t;
float *d_V1, *d_V2, *d_F;
float *d_x, *d_y;
float *d_t;

void Allocate(){

	size_t size;
	size = Np*Np*sizeof(float);

	V1 = (float*)malloc(size);
	V2 = (float*)malloc(size);
	F = (float*)malloc(size);
	ue = (float*)malloc(size);
	hipMalloc((void**)&d_V1, size);
	hipMalloc((void**)&d_V2, size);
	hipMalloc((void**)&d_F, size);


	size = Np*sizeof(float);

	x = (float*)malloc(size);
	y = (float*)malloc(size);
	hipMalloc((void**)&d_x, size);
	hipMalloc((void**)&d_y, size);

	size = sizeof(float);
	t = (float*)malloc(size);
        hipMalloc((void**)&d_t, size);
}

void Save_Result(){

	FILE *pFile;
	int i,j;
	int index;
	int n;
	n = Np;
	pFile = fopen("V1.txt","w+");
	// Save the matrix V1
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			index = i*n + j;
			fprintf(pFile, "%g", V1[index]);
			if (j == (n-1)) {
				fprintf(pFile, "\n");
			}else{
				fprintf(pFile, "\t");
			}
		}
	}
	fclose(pFile);
}

void Free(){
	free(V1);free(V2);free(F);free(ue);
	free(x);free(y);
	free(t);
	hipFree(d_V1);hipFree(d_V2);hipFree(d_F);
	hipFree(d_x);hipFree(d_y);
	hipFree(d_t);
}

void Send_to_Device(){
	hipError_t Error;
	size_t size;
        size = Np*Np*sizeof(float);

	Error = hipMemcpy(d_V1, V1, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy V1->d_V1) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy F->d_F) = %s\n",hipGetErrorString(Error));

	size = Np*sizeof(float);
        Error = hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy x->d_x) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy y->d_y) = %s\n",hipGetErrorString(Error));

	size = 1*sizeof(float);
        Error = hipMemcpy(d_t, t, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy t->d_t) = %s\n",hipGetErrorString(Error));
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = Np*Np*sizeof(float);

	Error = hipMemcpy(V1, d_V1, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V1->V1) = %s\n",hipGetErrorString(Error));

	size = 1*sizeof(float);
        Error = hipMemcpy(t, d_t, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_t->t) = %s\n",hipGetErrorString(Error));
}
