#include "hip/hip_runtime.h"
#include "head.h"

#define block_dim 32

extern float *d_F, *d_x, *d_y;
extern float *d_V1, *d_V2;
extern float *d_t;

__global__ void GPU_fsource(float *d_V1, float *d_t, float *d_x, float *d_y, float *d_F){

        int k = blockDim.x * blockIdx.x + threadIdx.x;
        int i, j;

        if(k<Np*Np){
		i = k/Np;
                j = k%Np;
                d_F[i*Np+j] = exp(-2.0*d_t[0])*cos(M_PI*d_x[i])*
                                                cos(M_PI*d_y[j])*(2.0*M_PI*M_PI)
                                                        -2.0*d_V1[i*Np+j];
        }
}

__global__ void forward(float *d_V1, float *d_V2, float *d_F){
	int y = gridDim.x * blockDim.x * (threadIdx.y + blockIdx.y * blockDim.y);
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int id = x + y;
	int newid = id - (threadIdx.y + blockIdx.y * blockDim.y)*(gridDim.x * blockDim.x - Np);
	int j = threadIdx.x;
	int i = threadIdx.y;
	int a = newid/Np;
	int b = newid%Np;

	__shared__ float V1[block_dim][block_dim];

	V1[i][j] = d_V1[newid];
	__syncthreads();

	if (newid<Np*Np){

		if(i>0&&i<(block_dim-1)&&j>0&&j<(block_dim-1)&&a>0&&a<Np-1&&b>0&&b<Np-1){
			d_V2[newid] = V1[i][j]+dt/h1*(V1[i-1][j]+V1[i+1][j]+V1[i][j-1]+V1[i][j+1]
					-4.0*V1[i][j])+dt*d_F[newid];
		}
		else if(a>0&&a<Np-1&&b>0&&b<Np-1){
			d_V2[a*Np+b] = d_V1[a*Np+b] + dt/h1*(d_V1[(a+1)*Np+b]+d_V1[(a-1)*Np+b]
					+d_V1[a*Np+b+1]+d_V1[a*Np+b-1]-4.0*d_V1[a*Np+b])
						+dt*d_F[a*Np+b];
		}
	}
}

__global__ void forward_bound(float *d_V1, float *d_V2, float *d_F){
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i;

	if(k<(Np-2)){
		i = k+1;
		d_V2[0*Np+i] = d_V1[0*Np+i] + dt/h1*
                                (d_V1[(0+1)*Np+i]+d_V1[(Np-1)*Np+i]+d_V1[0*Np+i+1]+d_V1[0*Np+i-1]
                                        -4*d_V1[0*Np+i]) + dt*d_F[0*Np+i];
		d_V2[(Np-1)*Np+i] = d_V1[(Np-1)*Np+i] + dt/h1*
                                (d_V1[0*Np+i]+d_V1[(Np-1-1)*Np+i]+d_V1[(Np-1)*Np+i+1]+d_V1[(Np-1)*Np+i-1]
                                        -4*d_V1[(Np-1)*Np+i]) + dt*d_F[(Np-1)*Np+i];
		d_V2[i*Np+0] = d_V1[i*Np+0] + dt/h1*
                                (d_V1[(i+1)*Np+0]+d_V1[(i-1)*Np+0]+d_V1[i*Np+0+1]+d_V1[i*Np+Np-1]
                                        -4*d_V1[i*Np+0]) + dt*d_F[i*Np+0];
		d_V2[i*Np+(Np-1)] = d_V1[i*Np+(Np-1)] + dt/h1*
                                (d_V1[(i+1)*Np+(Np-1)]+d_V1[(i-1)*Np+(Np-1)]+d_V1[i*Np+(Np-1)-1]+d_V1[i*Np+0]
                                        -4*d_V1[i*Np+(Np-1)]) + dt*d_F[i*Np+(Np-1)];
	}
	if(k==(Np-2)){
		d_V2[0*Np+0] = d_V1[0*Np+0] + dt/h1*
                                (d_V1[(0+1)*Np+0]+d_V1[(Np-1)*Np+0]+d_V1[0*Np+0+1]+d_V1[0*Np+Np-1]
					-4*d_V1[0*Np+0]) + dt*d_F[0*Np+0];
		d_V2[(Np-1)*Np+0] = d_V1[(Np-1)*Np+0] + dt/h1*
                                (d_V1[0*Np+0]+d_V1[(Np-1-1)*Np+0]+d_V1[(Np-1)*Np+0+1]+d_V1[(Np-1)*Np+Np-1]
                                        -4*d_V1[(Np-1)*Np+0]) + dt*d_F[(Np-1)*Np+0];
		d_V2[0*Np+(Np-1)] = d_V1[0*Np+(Np-1)] + dt/h1*
                                (d_V1[(0+1)*Np+(Np-1)]+d_V1[(Np-1)*Np+(Np-1)]+d_V1[0*Np+0]+d_V1[0*Np+(Np-1)-1]
                                        -4*d_V1[0*Np+(Np-1)]) + dt*d_F[0*Np+(Np-1)];
		d_V2[(Np-1)*Np+(Np-1)] = d_V1[(Np-1)*Np+(Np-1)] + dt/h1*
                                (d_V1[(Np-1-1)*Np+(Np-1)]+d_V1[0*Np+(Np-1)]+d_V1[(Np-1)*Np+0]+d_V1[(Np-1)*Np+(Np-1)-1]
                                        -4*d_V1[(Np-1)*Np+(Np-1)]) + dt*d_F[(Np-1)*Np+(Np-1)];
	}
}

__global__ void update_V1(float *d_V1, float *d_V2, float *d_t){
	int k = blockDim.x * blockIdx.x + threadIdx.x;

	if(k<(Np*Np)){
		d_V1[k] = d_V2[k];
	}

	if(k==0){
                d_t[0] = d_t[0]+dt;
        }

}

void forward_euler(){
        int tpb = block_dim;
        int bpg = (Np+tpb-1)/tpb;
	dim3 threads(tpb, tpb, 1);
        dim3 grid(bpg, bpg);
	forward<<<grid, threads>>>(d_V1, d_V2, d_F);

	tpb = 256;
	bpg = ((Np-1)+tpb-1)/tpb;
	forward_bound<<<bpg, tpb>>>(d_V1, d_V2, d_F);
	bpg = (Np*Np+tpb-1)/tpb;
	update_V1<<<bpg, tpb>>>(d_V1, d_V2, d_t);

	bpg = (Np*Np+tpb-1)/tpb;
	GPU_fsource<<<bpg, tpb>>>(d_V1, d_t, d_x, d_y, d_F);
	//hipDeviceSynchronize();
}
