#include "hip/hip_runtime.h"
#include "head.h"

#define block_dim 32

extern float *d_F, *d_x, *d_y;
extern float *d_V1, *d_V2;
extern float *d_t;

__global__ void GPU_fsource(float *d_V2, float *d_t, float *d_x, float *d_y, float *d_F){
        int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i, j;

        if(k<Np*Np){
		i = k/Np;
		j = k%Np;
		d_F[i*Np+j] = exp(-2.0*d_t[0])*cos(M_PI*d_x[i])*
                                                cos(M_PI*d_y[j])*(2.0*M_PI*M_PI)
                                                        -2.0*d_V2[i*Np+j];
        }
}

__global__ void forward(float *d_V1, float *d_V2, float *d_F){
	int yid = threadIdx.y + blockIdx.y * (blockDim.y-2);
	int xid = threadIdx.x + blockIdx.x * (blockDim.x-2);
	int totalx = (gridDim.x-1) * (blockDim.x-2) + blockDim.x;
	int id = xid + totalx * yid;
	int V2id = (yid-1)* (totalx-2) + xid - 1;
	int j = threadIdx.x;
	int i = threadIdx.y;

	__shared__ float V1[block_dim][block_dim];
	V1[i][j] = d_V1[id];
	__syncthreads();

	if(i>0&&i<(block_dim-1)&&j>0&&j<(block_dim-1)){
		d_V2[V2id] = V1[i][j] + dt/h1*(V1[i-1][j]+V1[i+1][j]+V1[i][j-1]+V1[i][j+1]
                                        -4*V1[i][j]) + dt*d_F[V2id];
	}
}

__global__ void update_V1(float *d_V1, float *d_V2, float *d_t){
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i, j;

	if(k<Np*Np){
		i = k/Np;
		j = k%Np;
		d_V1[(i+1)*(Np+2)+j+1] = d_V2[i*Np+j];
	}

	if(k<Np){
		d_V1[k+1] = d_V2[(Np-1)*Np+k];
                d_V1[(Np+1)*(Np+2)+k+1] = d_V2[k];
                d_V1[(k+1)*(Np+2)] = d_V2[k*Np+Np-1];
                d_V1[(k+1)*(Np+2)+Np+1] = d_V2[k*Np];
	}

	if(k==0){
                d_t[0] = d_t[0]+dt;
        }

}

void forward_euler(){
	int tpb = 256;
        int bpg = (Np*Np+tpb-1)/tpb;

	update_V1<<<bpg, tpb>>>(d_V1, d_V2, d_t);

	int twodtpb = block_dim;
	int twodbpg = (Np/(twodtpb-2));
	dim3 threads(twodtpb, twodtpb, 1);
	dim3 grid(twodbpg, twodbpg);
	forward<<<grid, threads>>>(d_V1, d_V2, d_F);

	GPU_fsource<<<bpg, tpb>>>(d_V2, d_t, d_x, d_y, d_F);
	//hipDeviceSynchronize();
}
